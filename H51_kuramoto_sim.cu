/*
 ;
 Name        : kuramoto_islets.cu
 Author      : Gerardo J. Felix-Martinez
 Version     : 0.0
 Copyright   : Your copyright notice
 Description : Implementacion de modelo de Kuramoto para sincronizacion de celulas en los islotes
 
 Graficar en gnuplot: plot for [col=2:4] 'AngulosIslote.data' using 1:col with lines

 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>


// Parametros globales modelo
#define totalCelulas 514
#define maxVecinos 10 // maximo numero de vecinos por celula
double *Angulos_Device;
double *Angulos_Host;
#define PI 3.14159265358979323846
//#define beta 0.23


// Parametros CUDA
const int numBlocks = 30;
const int threadsPerBlock = 64;
__const__ int NUMHILOS = numBlocks * threadsPerBlock;

struct contactos{
    int id_vecino; // Identificador de vecino en el islote
    double gacople; 
};

struct redcelulas {
    int n_celula;
    int n_vecinos;
    int tipo_celula;
    int id_vecinos[maxVecinos];
    int ind_arreglo_tipo; //identificador en su poblacion
    struct contactos vecinos[maxVecinos];
    double frec;
    double theta;
    double termino_acople;
};

__device__ struct redcelulas islote[totalCelulas];

// Generador de numeros aleatorios con distribucion uniforme
double get_random() { return ((double)rand() / (double)RAND_MAX); }


// Runge Kutta 4to orden
__device__ double rk4(double (*f)(double, int), double h, double x, int i) {
    double k1 = h * f(x, i);
    double k2 = h * f(x + k1 / 2, i);
    double k3 = h * f(x + k2 / 2, i);
    double k4 = h * f(x + k3, i);
    return x + (k1 + 2 * k2 + 2 * k3 + k4) / 6;
}

__global__ void set_TipoCelula(int i, int tipo) {
    // Guarda el tipo de celula que es la celula i
    islote[i].tipo_celula = tipo;

}
__global__ void set_idVecino(int i, int index, int idVecino) {
    // Guarda el id del vecino index de la celula i
    islote[i].id_vecinos[index] = idVecino;

}
__global__ void set_numVecinos(int i, int numvecinos) {
    // Guarda el numero total de vecinos que tiene la celula i
    islote[i].n_vecinos = numvecinos;
}

__global__ void set_idVecinoStruct(int i, int j, int numvecinos){
    islote[i].vecinos[numvecinos].id_vecino = j;
}


// Se calcula termino de acople
__device__ void calcular_acople(int i){
    int j;
    double sum_acoples = 0.0;
    // j itera sobre el numero de vecinos de la celula i
    for (j = 0; j < maxVecinos; j++){
        if (islote[i].id_vecinos[j] >= 0.0){
            sum_acoples = sum_acoples + islote[i].vecinos[j].gacople * sin(islote[islote[i].id_vecinos[j]].theta - islote[i].theta);
            //sum_acoples = sum_acoples + sin(islote[islote[i].id_vecinos[j]].theta - islote[i].theta);
        }
    }
    islote[i].termino_acople = sum_acoples;
    //printf("Celula %i, acople = %f\n", i, islote[i].termino_acople);
}

// Ecuacion diferencial modelo de Kuramoto
__device__ double dthetadt(double theta0, int i){
    //return islote[i].frec + islote[i].termino_acople;
    return islote[i].frec + 1.0/totalCelulas * islote[i].termino_acople;
    //return islote[i].frec + kappa/totalCelulas * islote[i].termino_acople;
}

// Funcion para generar numeros aleatorios con distribucion normal
double rand_normal(double mean, double stddev) { //Box muller method
    static double n2 = 0.0;
    static int n2_cached = 0;
    if (!n2_cached) {
        double x, y, r;
        do {
            x = 2.0 * rand() / RAND_MAX - 1;
            y = 2.0 * rand() / RAND_MAX - 1;

            r = x * x + y * y;
        } while (r == 0.0 || r > 1.0);
        {
            double d = sqrt(-2.0 * log(r) / r);
            double n1 = x * d;
            n2 = y * d;
            double result = n1 * stddev + mean;
            n2_cached = 1;
            return result;
        }
    } else {
        n2_cached = 0;
        return n2 * stddev + mean;
    }
}


void cargarArchivo() {
    FILE *fp;
    int i = 0;
//  fp = fopen("conectividad_a_b_d.txt", "r");
    
fp = fopen("/home/gerardo/Documents/IsletLab/H51_all_contacts.txt" ,"r");

    char linea[1000000];
    while (fgets(linea, sizeof(linea), fp)) {
        int numvecinos = 0;
        char *p = strtok(linea, " ");
        int j = 0;
        while (p != NULL) {
            //printf( " %s\t", p );
            int a = atoi(p);
            if (i == j) {
                // guarda tipo de celula de la celula i
                set_TipoCelula<<<1, 1>>>(i, a);
                hipDeviceSynchronize();
                // Para guardar el id del vecino numvecinos j de la celula i
                // como hay interacciones autocrinas se guarda la celula como su 
                // propia vecina
                //set_idVecino<<<1, 1>>>(i, numvecinos, j);
                //cudaDeviceSynchronize();
                //set_idVecinoStruct<<<1, 1>>>(i, j, numvecinos);
                //cudaDeviceSynchronize();
                //numvecinos++;
                //printf("Soy %i, y soy %i \t", i, islote[i].tipo_celula);
                //printf("\n");
            } else {
                if (a == 1) {
                    set_idVecino<<<1, 1>>>(i, numvecinos, j);
                    //cudaDeviceSynchronize();
                    set_idVecinoStruct<<<1, 1>>>(i, j, numvecinos);
                    hipDeviceSynchronize();
                    numvecinos++;
                }
            }
            p = strtok(NULL, " ");
            j++;
        }
        // Guarda el numero total de vecinos de la celula i 
        set_numVecinos<<<1, 1>>>(i, numvecinos);
        i++;
    }
}


__global__ void imprimeredcelula() {
    int j, i;
    for (i = 0; i < totalCelulas; i++) {
        printf("Soy %d, con %d vecinos \t", i, islote[i].n_vecinos);
        for (j = 0; j < maxVecinos; j++) {
            printf("%d ", islote[i].id_vecinos[j]);
        }
        printf("\n");
        //fflush(stdout);
    }
}

__global__ void iniciaredcelula(int i) {
    int j;
    for (j = 0; j < maxVecinos; j++) {
        islote[i].id_vecinos[j] = -1;
    }
    islote[i].n_celula = 0;
    islote[i].n_vecinos = 0;
    //double deltabeta = 2/(totalCelulas-1);
    //islote[i].frec = 1.+beta*(-1.0+(double)i*(2./(totalCelulas-1.)));
    //islote[i].frec = 1./10.;
}

__global__ void set_gacople(int i, double gacople, int numvecinos) {
    // guarda parametro de acople entre celula i y sus vecinos
    islote[i].vecinos[numvecinos].gacople = gacople;
    //printf("Soy %i con gacople = %f\n", i, islote[i].vecinos[numvecinos].gacople);
    printf("%i,%i,%f\n", i, islote[i].id_vecinos[numvecinos],islote[i].vecinos[numvecinos].gacople);
}


__global__ void asignar_theta(int i, double theta){
    // para pasar angulos generados en host a GPU
    islote[i].theta = theta;
    printf("Soy %i con angulo %f\n", i, islote[i].theta);
}

__global__ void asignar_frec(int i, double frec){
    islote[i].frec = frec;
}

void init_theta(){
    // Inicializa angulos para cada celula en host
    double theta;
    double frec;


    for (int i=0; i<totalCelulas; i++){
theta = 2 * PI * get_random();
        //theta = (2*(double)i*PI)/(double)totalCelulas;
        asignar_theta<<<1,1>>>(i, theta);
        hipDeviceSynchronize();
         frec = 0.0016666666666666668;
        //frec = rand_normal(0.001, 0.005);
        asignar_frec<<<1,1>>>(i, frec);
        //printf("%f\n", theta);
    }

}


void cargarArchivoCoupling() {
    FILE *fp;
    int i = 0;
fp = fopen("/home/gerardo/Documents/IsletLab/H51_Kmat.txt", "r");
    char linea[1000000];
    while (fgets(linea, sizeof(linea), fp)) {
        int numvecinos = 0;
        char *p = strtok(linea, " ");
        int j = 0;
        while (p != NULL) {
            //printf( " %s\t", p );
            double a = atof(p);
                if(a!=0.0){
                //if (a == 1) {
                    set_gacople<<<1, 1>>>(i, a, numvecinos);
                    hipDeviceSynchronize();
                    //islote[i].vecinos[numvecinos].gacople = a;
                    //printf("gcoup = %f\n", islote[i].vecinos[numvecinos].gacople);
                    numvecinos++;
                }
    
            p = strtok(NULL, " ");
            j++;
        }
        //islote[i].n_vecinos = numvecinos;
        i++;
    }
}


__global__ void calcula_InfluenciaVecinas() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
            + gridDim.x * gridDim.y * blockIdx.z;
    int tid = blockId * (blockDim.x * blockDim.y * blockDim.z)
            + (threadIdx.z * (blockDim.x * blockDim.y))
            + (threadIdx.y * blockDim.x) + threadIdx.x;
    int i = tid;

    while (i < totalCelulas) {
        calcular_acople(i);
        i = i + NUMHILOS;
    }

}

__device__ void resolver(int i, double dt, double (*mn)(double (*f)(double, int), double h, double xi,int i)){
    islote[i].theta = mn(dthetadt, dt, islote[i].theta, i);
}


__global__ void actualizaParametros(double dt) {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
            + gridDim.x * gridDim.y * blockIdx.z;
    int tid = blockId * (blockDim.x * blockDim.y * blockDim.z)
            + (threadIdx.z * (blockDim.x * blockDim.y))
            + (threadIdx.y * blockDim.x) + threadIdx.x;
    int i = tid;

    while (i < totalCelulas) {
        resolver(i, dt, rk4);
        i = i + NUMHILOS;
    }
}



__global__ void getAngulos(double *angulos){
    int i;
    for (i = 0; i < totalCelulas; ++i) {
        angulos[i] = islote[i].theta;
    }
}



int main(void){
    // seed numeros aleatorios
    srand(time(NULL)); // randomize seed

    time_t begin = time(NULL);

    // Tiempo total de simulacion
double Tf = 20000.0;
    double t;
    // Paso de tiempo simulacion
double dt = 0.1;
    int indice = 0;

    // Archivos para guardar 
FILE *salidaAngulosIslote = fopen("/home/gerardo/Documents/IsletLab/H51_kuramoto_angles.data", "w");

    // Memoria
    hipMalloc((void**) &Angulos_Device, totalCelulas * sizeof(double));
    Angulos_Host = (double*) malloc(totalCelulas * sizeof(double));

    for (int cell = 0; cell < totalCelulas; cell++) {
        iniciaredcelula<<<1, 1>>>(cell);
        hipDeviceSynchronize();
    }
    
    // Carga matriz de adyacencias y guarda tipo de celulas
    // y vecinos de cada celula
    cargarArchivo();

    // Imprime numero de celula, numero de vecinos e id de vecinos
    imprimeredcelula<<<1,1>>>();
    hipDeviceSynchronize();

    // Carga los parametros K de los osciladores
    // En islote[i].vecinos[numvecinos].gacople
    // queda guardada la conductancia entre la celula i y 
    // el vecino islote[i].id_vecinos[numvecinos],
    cargarArchivoCoupling();

    // Inicializa angulos (theta)
    init_theta();

    for (t = 0; t < Tf;) {
if (indice % 500 == 0){
            getAngulos<<<1,1>>>(Angulos_Device);
            hipDeviceSynchronize();

            hipMemcpy(Angulos_Host, Angulos_Device, totalCelulas * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            printf("%lf\n",t); fflush(stdout);
            fprintf(salidaAngulosIslote, "%lf\t", t);
            for (int i = 0; i < totalCelulas; ++i) {
                if (i == totalCelulas - 1) {
                    fprintf(salidaAngulosIslote, "%lf\n", Angulos_Host[i]);
                } else {
                    fprintf(salidaAngulosIslote, "%lf\t", Angulos_Host[i]);
                }
            }
        }

        calcula_InfluenciaVecinas<<<numBlocks, threadsPerBlock>>>();
        hipDeviceSynchronize();

        actualizaParametros<<<numBlocks, threadsPerBlock>>>(dt);
        hipDeviceSynchronize();

    
        indice++;
        t = t + dt;

    }
    time_t end = time(NULL);
    printf("Tiempo de ejecucion: %ld segundos\n", (end - begin));
    fflush(stdout);

    return EXIT_SUCCESS;
    
}
